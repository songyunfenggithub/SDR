#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "myDebug.h"
#include "CWaveData.h"
#include "CWaveFilter.h"
#include "CDemodulatorAM.h"

#include "cuda_AM_Filter.cuh"


FILTERCOREDATATYPE* d_AM_Filter_Core = NULL;
ADCDATATYPE* d_AM_SrcData = NULL;
FILTEDDATATYPE* d_AM_Filtted_Result = NULL;
FILTEDDATATYPE* d_AM_Demodulator_Result = NULL;

//FILTERCOREDATATYPE* h_Filter_Core = NULL;
ADCDATATYPE* h_AM_SrcData = NULL;
FILTEDDATATYPE* h_AM_Filtted_Result = NULL;
FILTEDDATATYPE* h_AM_Demodulator_Result = NULL;

size_t cuda_AM_thread_size;
size_t cuda_AM_filter_result_step_size;
size_t cuda_AM_demodulator_result_size;
size_t cuda_AM_src_step_size;

__global__ void
cuda_AM_Filter(const ADCDATATYPE* src, const FILTERCOREDATATYPE* core, FILTEDDATATYPE* filtted_result, FILTEDDATATYPE* demodulator_result, 
	int stage, unsigned int corelen)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	__shared__ float cache[CUDA_FILTER_BUFF_SRC_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT];
	unsigned int cache_length_mask = (CUDA_FILTER_BUFF_SRC_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT) - 1;
	unsigned int cache_stage_length = CUDA_FILTER_BUFF_STEP_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT;
	unsigned int cache_stage_step = stage * cache_stage_length;

	unsigned int src_stage_step = stage * CUDA_FILTER_BUFF_STEP_LENGTH;

	unsigned int index = blockIdx.x;
	unsigned int i_stage_step = cache_stage_step + index - corelen;

	cache[cache_stage_step + index] = src[src_stage_step + (index << DEMODULATOR_AM_DECIMATION_FACTOR_BIT)];

	//���߳̿��е��߳̽���ͬ��
	__syncthreads();

	filtted_result[index] = 0;
	for (int i = 0; i < corelen; i++) {
		filtted_result[cache_stage_step + index] += cache[(i_stage_step + i) & cache_length_mask] * core[i];
	}
	
	//���߳̿��е��߳̽���ͬ��
	__syncthreads();
	
	unsigned int n = 1 << DEMODULATOR_AM_FILTER_SAMPLERATE_OFFSET_BIT;
	unsigned int mask = n - 1;
	unsigned int demodulator_result_index;
	float d;
	if ((index & mask) == 0) {
		demodulator_result_index = index >> DEMODULATOR_AM_FILTER_SAMPLERATE_OFFSET_BIT;
		demodulator_result[demodulator_result_index] = 0;
		for (int i = 0; i < n; i++) {
			if((d = filtted_result[cache_stage_step + index + i]) > 0) demodulator_result[demodulator_result_index] += d;
		}
	}
}

void Cuda_AM_ReInitFilterCore(CWaveFilter::PFILTERINFO pFilterInfo)
{
	hipError_t err = hipSuccess;

	if (d_AM_Filter_Core != NULL) {
		err = hipFree(d_AM_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_AM_Filter_Core failed!\r\n");
			//goto Error;
		}
	}
	//if(h_Filter_Core != NULL)free(h_Filter_Core);

	size_t filter_core_size = pFilterInfo->CoreLength * sizeof(FILTERCOREDATATYPE);
	// Allocate the host input vector Core
	//h_Filter_Core = (FILTERCOREDATATYPE*)malloc(filter_core_size);

	//memcpy(h_Filter_Core, clsWaveFilter.FilterCore, filter_core_size);

	// Allocate the device input vector Filter_Core
	err = hipMalloc((void**)&d_AM_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_AM_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipMemcpy(d_AM_Filter_Core, pFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_AM_Filter_Core failed!\r\n");
		//goto Error;
	}
	if (d_AM_Demodulator_Result != NULL) {
		err = hipFree(d_AM_Demodulator_Result);
		if (err != hipSuccess) {
			printf("hipFree d_AM_Demodulator_Result failed!\r\n");
			//goto Error;
		}
	}
	cuda_AM_demodulator_result_size = (clsWaveData.AdcSampleRate >> (DEMODULATOR_AM_FILTER_SAMPLERATE_OFFSET_BIT + DEMODULATOR_AM_DECIMATION_FACTOR_BIT)) * sizeof(ADCDATATYPE);
	err = hipMalloc((void**)&d_AM_Demodulator_Result, cuda_AM_demodulator_result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_AM_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	printf("Cuda ReInit AM Filter Core Done.\r\n");

}

void cuda_AM_getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);
	printf("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	printf("max thread num: %d\n", prop.maxThreadsPerBlock);
	printf("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void Cuda_AM_Filter_Init(void)
{

	cuda_AM_getThreadNum();

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	cuda_AM_src_step_size = CUDA_FILTER_BUFF_STEP_LENGTH * sizeof(ADCDATATYPE);
	printf("Cuda_Init [Vector addition of %d CUDA_FILTER_BUFF_SRC_LENGTH]\n", CUDA_FILTER_BUFF_SRC_LENGTH);

	// Allocate the host input vector A
	//h_AM_SrcData = (ADCDATATYPE*)malloc(src_data_size);

	cuda_AM_thread_size = CUDA_FILTER_BUFF_STEP_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT;
	cuda_AM_filter_result_step_size = CUDA_FILTER_BUFF_STEP_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT * sizeof(FILTEDDATATYPE);
	// Allocate the host output vector Result
	//float* h_AM_Result = (float*)malloc(cuda_AM_result_size);

	// Allocate the device input vector A
	size_t src_data_size = CUDA_FILTER_BUFF_SRC_LENGTH * sizeof(ADCDATATYPE);
	err = hipMalloc((void**)&d_AM_SrcData, src_data_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_AM_SrcData failed!\r\n");
		//goto Error;
	}
	// Allocate the device output vector C
	cuda_AM_filter_result_step_size = (CUDA_FILTER_BUFF_STEP_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT) * sizeof(ADCDATATYPE);
	size_t filter_result_size = (CUDA_FILTER_BUFF_SRC_LENGTH >> DEMODULATOR_AM_DECIMATION_FACTOR_BIT) * sizeof(ADCDATATYPE);
	err = hipMalloc((void**)&d_AM_Filtted_Result, filter_result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_AM_Filtted_Result failed!\r\n");
		//goto Error;
	}

	Cuda_AM_ReInitFilterCore(clsDemodulatorAm.pFilterInfo);

	printf("Cuda Demodulator AM Init Done.\r\n");
}

void Cuda_AM_Filter_UnInit(void)
{

	while (clsWaveFilter.cud_Filter_exit == false);

	hipError_t err = hipSuccess;

	// Free device global memory
	err = hipFree(d_AM_SrcData);
	if (err != hipSuccess) {
		printf("hipFree d_AM_SrcData failed!\r\n");
		//goto Error;
	}
	err = hipFree(d_AM_Filter_Core);
	if (err != hipSuccess) {
		printf("hipFree d_AM_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipFree(d_AM_Filtted_Result);
	if (err != hipSuccess) {
		printf("hipFree d_AM_Filtted_Result failed!\r\n");
		//goto Error;
	}
	err = hipFree(d_AM_Demodulator_Result);
	if (err != hipSuccess) {
		printf("hipFree d_AM_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	// Free host memory
	//free(h_Filter_Core);

	printf("Cuda_Filter_Closed.\r\n");
}

void cuda_AM_Filtting(void)
{
	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = cuda_AM_thread_size;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;

	h_AM_SrcData = (ADCDATATYPE*)clsWaveData.AdcBuff + clsWaveData.FilttedPos;
	//printf("stage:%d\r\n", stage);
	err = hipMemcpy((char*)d_AM_SrcData + stage * cuda_AM_src_step_size, h_AM_SrcData, cuda_AM_src_step_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_AM_SrcData failed!\r\n");
	}

	WaitForSingleObject(clsWaveFilter.hCoreMutex, INFINITE);

	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	cuda_AM_Filter << <threadsPerBlock, blocksPerGrid >> > (d_AM_SrcData, d_AM_Filter_Core, h_AM_Filtted_Result, d_AM_Demodulator_Result, 
		stage, clsDemodulatorAm.pFilterInfo->CoreLength);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}

	ReleaseMutex(clsWaveFilter.hCoreMutex);

	h_AM_Filtted_Result = (FILTEDDATATYPE*)clsDemodulatorAm.FilttedBuff + clsDemodulatorAm.FilttedPos;
	err = hipMemcpy(h_AM_Filtted_Result, d_AM_Filtted_Result + stage * cuda_AM_filter_result_step_size, cuda_AM_filter_result_step_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_AM_Filtted_Result failed!\r\n");
	}
	clsDemodulatorAm.FilttedPos += CUDA_FILTER_BUFF_STEP_LENGTH;
	clsDemodulatorAm.FilttedPos &= DEMODULATOR_AM_FILTTED_BUFF_LENGTH_MASK;

	h_AM_Demodulator_Result = (FILTEDDATATYPE*)clsDemodulatorAm.AMBuff + clsDemodulatorAm.AMPos;
	err = hipMemcpy(h_AM_Demodulator_Result, d_AM_Demodulator_Result, cuda_AM_demodulator_result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_AM_Demodulator_Result failed!\r\n");
	}
	clsDemodulatorAm.AMPos += cuda_AM_demodulator_result_size / sizeof(FILTEDDATATYPE);
	clsDemodulatorAm.AMPos &= DEMODULATOR_AM_BUFF_LENGTH_MASK;

	stage++;
	stage &= 0x3;
}