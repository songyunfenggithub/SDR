#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <stdlib.h>
#include <time.h>
#include <Windows.h>

#define _USE_MATH_DEFINES
#include <math.h>

#include "Debug.h"
#include "CData.h"
#include "cuda_CFFT.cuh"

using namespace METHOD;

#define CUFFT_CALL( call )                                                                                             \
    {                                                                                                                  \
        auto status = static_cast<hipfftResult>( call );                                                                \
        if ( status != HIPFFT_SUCCESS )                                                                                 \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUFFT call \"%s\" in line %d of file %s failed "                                          \
                     "with "                                                                                           \
                     "code (%d).\n",                                                                                   \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     status );                                                                                         \
    }

cuda_CFFT::cuda_CFFT()
{

}

cuda_CFFT::~cuda_CFFT()
{
    cuda_FFT_UnInit();
}

void cuda_CFFT::cuda_FFT(void)
{
    float cost, s;
    s = GetTickCount();
    CUFFT_CALL(hipMemcpy(cuda_FFT_d_fftData, cuda_FFT_CompiData, FFTInfo->FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));// copy data from host to device
    
    //WaitForSingleObject(cuda_FFT_hMutexBuff, INFINITE);
    CUFFT_CALL(hipfftExecZ2Z(cuda_FFT_fft_plan, (hipfftDoubleComplex*)cuda_FFT_d_fftData, (hipfftDoubleComplex*)cuda_FFT_d_outfftData, HIPFFT_FORWARD));//execute
    CUFFT_CALL(hipDeviceSynchronize());//wait to be done
    //ReleaseMutex(cuda_FFT_hMutexBuff);

    CUFFT_CALL(hipMemcpy(cuda_FFT_CompoData, cuda_FFT_d_outfftData, FFTInfo->FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));// copy the result from device to host
    //DbgMsg("Time of cudaFFT: %fms\r\n", GetTickCount() - s);
}

void cuda_CFFT::cuda_FFT_Init(CData* data)
{
    cuda_FFT_UnInit();

    Data = data;

    if (cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    cuda_FFT_CompiData = (hipfftDoubleComplex*)malloc(FFTInfo->FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    cuda_FFT_CompoData = (hipfftDoubleComplex*)malloc(FFTInfo->FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    hipMalloc((void**)&cuda_FFT_d_fftData, FFTInfo->FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    hipMalloc((void**)&cuda_FFT_d_outfftData, FFTInfo->FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);
    hipfftPlan1d(&cuda_FFT_fft_plan, FFTInfo->FFTSize, HIPFFT_Z2Z, 1);//declaration

    FFTMaxValue = Get_FFT_Max_Value();
}

void cuda_CFFT::cuda_FFT_UnInit(void)
{
    if (cuda_FFT_CompiData != NULL) {
        free(cuda_FFT_CompiData);
        cuda_FFT_CompiData = NULL;
    }
    if (cuda_FFT_CompoData != NULL) {
        free(cuda_FFT_CompoData);
        cuda_FFT_CompoData = NULL;
    }
    if (cuda_FFT_d_fftData != NULL) {
        hipFree(cuda_FFT_d_fftData);
        cuda_FFT_d_fftData = NULL;
    }
    if (cuda_FFT_d_outfftData != NULL) {
        hipFree(cuda_FFT_d_outfftData);
        cuda_FFT_d_outfftData = NULL;
    }
    if (cuda_FFT_fft_plan != 0) {
        hipfftDestroy(cuda_FFT_fft_plan);
        cuda_FFT_fft_plan = 0;
    }
    DbgMsg("Cuda_CFFT_Closed.\r\n");
}

void cuda_CFFT::cuda_FFT(UINT pos)
{
    cuda_FFT_Prepare_Data(pos);
    cuda_FFT();
}

void cuda_CFFT::cuda_FFT_Prepare_Data(UINT pos)
{
    int i;
    CData* cData = (CData*)Data;
    memset(cuda_FFT_CompiData, 0, FFTInfo->FFTSize * sizeof(hipfftDoubleComplex));
    switch(cData->DataType)
    {
    case BUFF_DATA_TYPE::char_type:
    {
        char* buff = (char*)cData->Buff;
        for (i = 0; i < FFTInfo->FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = buff[pos & cData->Mask];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    break;
    case BUFF_DATA_TYPE::short_type:
    {
        short* buff = (short*)cData->Buff;
        for (i = 0; i < FFTInfo->FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = buff[pos & cData->Mask];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    break;
    case BUFF_DATA_TYPE::float_type:
    {
        float* buff = (float*)cData->Buff;
        for (i = 0; i < FFTInfo->FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = buff[pos & cData->Mask];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    break;
    }
}

void cuda_CFFT::cuda_FFT_Prepare_Data_for_MaxValue(double* buff)
{
    for (int i = 0; i < FFTInfo->FFTSize; i++) {
        cuda_FFT_CompiData[i].x = (double)buff[i];
        cuda_FFT_CompiData[i].y = 0;
    }
}

double cuda_CFFT::Get_FFT_Max_Value(void)
{
    double* buff = new double[FFTInfo->FFTSize];
    int i;
    double maxd = 0;
    UINT64 max = ((UINT64)1 << (Data->DataBits -1)) - 1;
    for (i = 0; i < FFTInfo->FFTSize; i++) buff[i] = (double)max * sin(2 * M_PI * i / FFTInfo->FFTSize);
    cuda_FFT_Prepare_Data_for_MaxValue(buff);
    cuda_FFT();
    int f = 1;
    maxd = sqrt(cuda_FFT_CompoData[f].x * cuda_FFT_CompoData[f].x + cuda_FFT_CompoData[f].y * cuda_FFT_CompoData[f].y);
    DbgMsg("maxvalue:%d, %lf\n", max, maxd);
    free(buff);
    return maxd;
}