#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <stdlib.h>
#include <time.h>
#include <Windows.h>

#include "CFFT.h"

#include "cuda_CFFT.cuh"

#define CUFFT_CALL( call )                                                                                             \
    {                                                                                                                  \
        auto status = static_cast<hipfftResult>( call );                                                                \
        if ( status != HIPFFT_SUCCESS )                                                                                 \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUFFT call \"%s\" in line %d of file %s failed "                                          \
                     "with "                                                                                           \
                     "code (%d).\n",                                                                                   \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     status );                                                                                         \
    }

void cuda_CFFT::cuda_FFT(void)
{
    float cost, s;
    s = GetTickCount();
    CUFFT_CALL(hipMemcpy(cuda_FFT_d_fftData, cuda_FFT_CompiData, FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));// copy data from host to device
    
    //WaitForSingleObject(cuda_FFT_hMutexBuff, INFINITE);
    CUFFT_CALL(hipfftExecZ2Z(cuda_FFT_fft_plan, (hipfftDoubleComplex*)cuda_FFT_d_fftData, (hipfftDoubleComplex*)cuda_FFT_d_outfftData, HIPFFT_FORWARD));//execute
    CUFFT_CALL(hipDeviceSynchronize());//wait to be done
    //ReleaseMutex(cuda_FFT_hMutexBuff);

    CUFFT_CALL(hipMemcpy(cuda_FFT_CompoData, cuda_FFT_d_outfftData, FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));// copy the result from device to host
    //printf("Time of cudaFFT: %fms\r\n", GetTickCount() - s);
}

void cuda_CFFT::cuda_FFT_Init(CFFT *fft)
{
    this->fft = fft;
    FFTSize = fft->FFTSize;
    FFTStep = fft->FFTStep;

    if (cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    cuda_FFT_CompiData = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    cuda_FFT_CompoData = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    hipMalloc((void**)&cuda_FFT_d_fftData, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    hipMalloc((void**)&cuda_FFT_d_outfftData, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);
    hipfftPlan1d(&cuda_FFT_fft_plan, FFTSize, HIPFFT_Z2Z, 1);//declaration
}

void cuda_CFFT::cuda_FFT_UnInit(void)
{
    while (fft->FFTT_hread_Exit == false);

    if (cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    if (cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    if (cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);
    printf("Cuda_CFFT_Closed.\r\n");
}

void cuda_CFFT::cuda_FFT(void* Buff, BUFF_DATA_TYPE type, UINT pos, UINT mask)
{
    cuda_FFT_Prepare_Data(Buff, type, pos, mask);
    cuda_FFT();
}

void cuda_CFFT::cuda_FFT_Prepare_Data(void* Buff, BUFF_DATA_TYPE type, UINT pos, UINT mask)
{
    int i;
    memset(cuda_FFT_CompiData, 0, FFTSize * sizeof(hipfftDoubleComplex));
    switch(type)
    {
    case BUFF_DATA_TYPE::char_type:
    {
        char* buff = (char*)Buff;
        for (i = 0; i < FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = buff[pos & mask];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    break;
    case BUFF_DATA_TYPE::short_type:
    {
        short* buff = (short*)Buff;
        for (i = 0; i < FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = buff[pos & mask];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    break;
    case BUFF_DATA_TYPE::float_type:
    {
        float* buff = (float*)Buff;
        for (i = 0; i < FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = buff[pos & mask];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    break;

    }
}

void cuda_CFFT::cuda_FFT_Prepare_Data_for_MaxValue(double* buff)
{
    for (int i = 0; i < FFTSize; i++) {
        cuda_FFT_CompiData[i].x = (double)buff[i];
        cuda_FFT_CompiData[i].y = 0;
    }
}
