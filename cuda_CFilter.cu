#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "public.h"
#include "myDebug.h"
#include "CData.h"
#include "CFilter.h"

#include "cuda_CFilter.cuh"

using namespace METHOD;

cuda_CFilter clscudaMainFilter;
cuda_CFilter clscudaAudioFilter;

__global__ void
cuda_Filter(const ADC_DATA_TYPE* src, const FILTER_CORE_DATA_TYPE* core, FILTTED_DATA_TYPE* result, int stage, unsigned int corelen, unsigned int srclen)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * (srclen >> 2);
	unsigned int istep = step + index - corelen;
	unsigned int srcmask = srclen - 1;
	result[index] = 0;
	for (int n = 0; n < corelen; n++) {
		result[index] += src[(istep + n) & srcmask] * core[n];
	}
}

__global__ void
cuda_Filter_short(const short* src, const FILTER_CORE_DATA_TYPE* core, FILTTED_DATA_TYPE* result, int stage, unsigned int corelen, unsigned int srclen)
{
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * (srclen >> 2);
	unsigned int istep = step + index - corelen;
	unsigned int srcmask = srclen - 1;
	result[index] = 0;
	for (int n = 0; n < corelen; n++) {
		result[index] += src[(istep + n) & srcmask] * core[n];
	}
}

__global__ void
cuda_Filter_float(const float* src, const FILTER_CORE_DATA_TYPE* core, FILTTED_DATA_TYPE* result, int stage, unsigned int corelen, unsigned int srclen)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * (srclen >> 2);
	unsigned int istep = step + index - corelen;
	unsigned int srcmask = srclen - 1;
	result[index] = 0;
	for (int n = 0; n < corelen; n++) {
		result[index] += src[(istep + n) & srcmask] * core[n];
	}
}

void cuda_CFilter::getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);

	printf("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	printf("max thread num: %d\n", prop.maxThreadsPerBlock);
	printf("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void cuda_CFilter::Init(CFilter::PFILTER_INFO pFilterInfo, CData* srcData, CData* targetData, UINT srcLen)
{
	rootFilterInfo = pFilterInfo;
	SrcData = srcData;
	TargetData = targetData;
	SrcLen = srcLen;

	UnInit();

	getThreadNum();

	hipError_t err = hipSuccess;

	int numElements = srcLen;
	size_t src_data_size = numElements * srcData->SizeOfType;
	src_step_size = (srcLen >> 2) * srcData->SizeOfType;
	printf("Cuda_Init [Vector addition of %d elements]\n", numElements);

	thread_size = (srcLen >> 2);
	result_size = thread_size * targetData->SizeOfType;

	err = hipMalloc((void**)&d_SrcData, src_data_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_SrcData failed!\r\n");
	}
	err = hipMalloc((void**)&d_Result, result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Result failed!\r\n");
	}
	
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
		}
	}
	size_t filter_core_size = pFilterInfo->CoreLength * sizeof(FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Filter_Core failed!\r\n");
	}
	err = hipMemcpy(d_Filter_Core, pFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_Filter_Core failed!\r\n");
	}

	printf("Cuda Init Done.\r\n");
}

void cuda_CFilter::UnInit(void)
{

	hipError_t err = hipSuccess;
	
	// Free device global memory
	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			printf("hipFree d_SrcData failed!\r\n");
		}
		d_SrcData = NULL;
	}
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
		}
		d_Filter_Core = NULL;
	}
	if (d_Result != NULL) {
		err = hipFree(d_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Result failed!\r\n");
		}
		d_Result = NULL;
	}
	// Free host memory
	//free(h_Filter_Core);

	DbgMsg("cuda_CFilter UnInit Closed.\r\n");
}

void cuda_CFilter::Filtting(void)
{
	WaitForSingleObject(clsMainFilter.hCoreMutex, INFINITE);

	CData* src_Data = (CData*)SrcData;
	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = thread_size;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;
	
	h_SrcData = (char*)src_Data->Buff + src_Data->SizeOfType * src_Data->ProcessPos;
	err = hipMemcpy((char*)d_SrcData + stage * src_step_size, h_SrcData, src_step_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_SrcData failed!\r\n");
	}
	
	switch (src_Data->DataType) {
	case short_type:
	{
		cuda_Filter_short << <threadsPerBlock, blocksPerGrid >> > ((short*)d_SrcData, d_Filter_Core, d_Result, stage,
			rootFilterInfo->CoreLength, SrcLen);
	}
	break;
	case float_type:
	{
		cuda_Filter_float << <threadsPerBlock, blocksPerGrid >> > ((float*)d_SrcData, d_Filter_Core, d_Result, stage,
			rootFilterInfo->CoreLength, SrcLen);
	}
	break;
	}
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}
	
	CData* target_Data = (CData*)TargetData;
	h_Result = (FILTTED_DATA_TYPE*)target_Data->Buff + target_Data->Pos;
	err = hipMemcpy(h_Result, d_Result, result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_Result failed!\r\n");
	}
	src_Data->ProcessPos += (SrcLen >> 2);
	src_Data->ProcessPos &= src_Data->Mask;
	target_Data->Pos = src_Data->ProcessPos;

	stage++;
	stage &= 0x3;
	//printf("Cuda Filtting Done.\r\n");

	ReleaseMutex(clsMainFilter.hCoreMutex);
}