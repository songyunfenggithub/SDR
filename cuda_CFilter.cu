#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "myDebug.h"
#include "CData.h"
#include "CWaveFilter.h"

#include "cuda_CFilter.cuh"

cuda_CFilter clscudaFilter;

__global__ void
cuda_Filter(const ADCDATATYPE* src, const FILTER_CORE_DATA_TYPE* core, FILTEDDATATYPE* result, int stage, unsigned int corelen)
{
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * CUDA_FILTER_BUFF_STEP_LENGTH;
	unsigned int istep = step + index - corelen;
	result[index] = 0;
	for (int n = 0; n < corelen; n++) {
		result[index] += src[(istep + n) & CUDA_FILTER_BUFF_SRC_LENGTH_MASK] * core[n];
	}
}

void cuda_CFilter::getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);

	printf("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	printf("max thread num: %d\n", prop.maxThreadsPerBlock);
	printf("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void cuda_CFilter::Init(CWaveFilter::PFILTERINFO pFilterInfo)
{
	rootFilterInfo = pFilterInfo;

	UnInit();

	getThreadNum();

	hipError_t err = hipSuccess;

	int numElements = CUDA_FILTER_BUFF_SRC_LENGTH;
	size_t src_data_size = numElements * sizeof(ADCDATATYPE);
	src_step_size = CUDA_FILTER_BUFF_STEP_LENGTH * sizeof(ADCDATATYPE);
	printf("Cuda_Init [Vector addition of %d elements]\n", numElements);

	thread_size = CUDA_FILTER_BUFF_STEP_LENGTH;
	result_size = thread_size * sizeof(FILTEDDATATYPE);

	err = hipMalloc((void**)&d_SrcData, src_data_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_SrcData failed!\r\n");
	}
	err = hipMalloc((void**)&d_Result, result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Result failed!\r\n");
	}
	
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
		}
	}
	size_t filter_core_size = pFilterInfo->CoreLength * sizeof(FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Filter_Core failed!\r\n");
	}
	err = hipMemcpy(d_Filter_Core, pFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_Filter_Core failed!\r\n");
	}

	printf("Cuda Init Done.\r\n");
}

void cuda_CFilter::UnInit(void)
{

	hipError_t err = hipSuccess;
	
	// Free device global memory
	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			printf("hipFree d_SrcData failed!\r\n");
		}
		d_SrcData = NULL;
	}
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
		}
		d_Filter_Core = NULL;
	}
	if (d_Result != NULL) {
		err = hipFree(d_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Result failed!\r\n");
		}
		d_Result = NULL;
	}
	// Free host memory
	//free(h_Filter_Core);

	DbgMsg("cuda_CFilter UnInit Closed.\r\n");
}

void cuda_CFilter::Filtting(void)
{
	WaitForSingleObject(clsWaveFilter.hCoreMutex, INFINITE);

	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = thread_size;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;

	h_SrcData = (ADCDATATYPE*)clsData.AdcBuff + clsData.FilttedPos;
	err = hipMemcpy((char*)d_SrcData + stage * src_step_size, h_SrcData, src_step_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_SrcData failed!\r\n");
	}

	cuda_Filter << <threadsPerBlock, blocksPerGrid >> > (d_SrcData, d_Filter_Core, d_Result, stage,
		rootFilterInfo->CoreLength);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}
	
	h_Result = (FILTEDDATATYPE*)clsData.FilttedBuff + clsData.FilttedBuffPos;
	err = hipMemcpy(h_Result, d_Result, result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_Result failed!\r\n");
	}
	clsData.FilttedPos += CUDA_FILTER_BUFF_STEP_LENGTH;
	clsData.FilttedPos &= DATA_BUFFER_MASK;
	clsData.FilttedBuffPos = clsData.FilttedPos;

	stage++;
	stage &= 0x3;
	//printf("Cuda Filtting Done.\r\n");

	ReleaseMutex(clsWaveFilter.hCoreMutex);
}