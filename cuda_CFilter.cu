#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "public.h"
#include "myDebug.h"
#include "CData.h"
#include "CFilter.h"

#include "cuda_CFilter.cuh"

using namespace METHOD;

cuda_CFilter clscudaMainFilter;
cuda_CFilter clscudaAudioFilter;

__global__ void
cuda_Filter(const ADC_DATA_TYPE* src, const FILTER_CORE_DATA_TYPE* core, FILTTED_DATA_TYPE* result, int stage, unsigned int corelen, unsigned int srclen, float scale)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * (srclen >> 2);
	unsigned int istep = step + index - corelen;
	unsigned int srcmask = srclen - 1;
	float d = 0;
	for (int n = 0; n < corelen; n++) {
		d += src[(istep + n) & srcmask] * core[n];
	}
	result[index] = d * scale;
}

__global__ void
cuda_Filter_short(const short* src, const FILTER_CORE_DATA_TYPE* core, FILTTED_DATA_TYPE* result, int stage, unsigned int corelen, unsigned int srclen, float scale)
{
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * (srclen >> 2);
	unsigned int istep = step + index - corelen;
	unsigned int srcmask = srclen - 1;
	float d = 0;
	for (int n = 0; n < corelen; n++) {
		d += src[(istep + n) & srcmask] * core[n];
	}
	result[index] = d * scale;
}

__global__ void
cuda_Filter_float(const float* src, const FILTER_CORE_DATA_TYPE* core, FILTTED_DATA_TYPE* result, int stage, unsigned int corelen, unsigned int srclen, float scale)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * (srclen >> 2);
	unsigned int istep = step + index - corelen;
	unsigned int srcmask = srclen - 1;
	float d = 0;
	for (int n = 0; n < corelen; n++) {
		d += src[(istep + n) & srcmask] * core[n];
	}
	result[index] = d * scale;
}

void cuda_CFilter::getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);

	printf("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	printf("max thread num: %d\n", prop.maxThreadsPerBlock);
	printf("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void cuda_CFilter::Init(CFilter* f)
{
	cFilter = f;
	rootFilterInfo = &f->rootFilterInfo1;
	SrcData = f->SrcData;
	TargetData = f->TargetData;
	SrcLen = f->FilterSrcLen;

	UnInit();

	getThreadNum();

	hipError_t err = hipSuccess;

	int numElements = SrcLen;
	size_t src_data_size = numElements * SrcData->SizeOfType;
	src_step_size = (SrcLen >> 2) * SrcData->SizeOfType;
	printf("Cuda_Init [Vector addition of %d elements]\n", numElements);

	thread_size = (SrcLen >> 2);
	result_size = thread_size * TargetData->SizeOfType;

	err = hipMalloc((void**)&d_SrcData, src_data_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_SrcData failed!\r\n");
	}
	err = hipMalloc((void**)&d_Result, result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Result failed!\r\n");
	}
	
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
		}
	}
	size_t filter_core_size = rootFilterInfo->CoreLength * sizeof(FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Filter_Core failed!\r\n");
	}
	err = hipMemcpy(d_Filter_Core, rootFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_Filter_Core failed!\r\n");
	}

	printf("Cuda Init Done.\r\n");
}

void cuda_CFilter::UnInit(void)
{

	hipError_t err = hipSuccess;
	
	// Free device global memory
	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			printf("hipFree d_SrcData failed!\r\n");
		}
		d_SrcData = NULL;
	}
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
		}
		d_Filter_Core = NULL;
	}
	if (d_Result != NULL) {
		err = hipFree(d_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Result failed!\r\n");
		}
		d_Result = NULL;
	}
	// Free host memory
	//free(h_Filter_Core);

	DbgMsg("cuda_CFilter UnInit Closed.\r\n");
}

void cuda_CFilter::Filtting(void)
{
	WaitForSingleObject(cFilter->hCoreMutex, INFINITE);

	if (cFilter->Cuda_Filter_N_New != cFilter->Cuda_Filter_N_Doing) {
		ReleaseMutex(cFilter->hCoreMutex);
		Sleep(100);
		return;
	}

	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = thread_size;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;
	
	h_SrcData = (char*)SrcData->Buff + SrcData->SizeOfType * SrcData->ProcessPos;
	err = hipMemcpy((char*)d_SrcData + stage * src_step_size, h_SrcData, src_step_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_SrcData failed!\r\n");
	}
	
	switch (SrcData->DataType) {
	case short_type:
	{
		cuda_Filter_short << <threadsPerBlock, blocksPerGrid >> > ((short*)d_SrcData, d_Filter_Core, d_Result, stage,
			rootFilterInfo->CoreLength, SrcLen, *cFilter->scale);
	}
	break;
	case float_type:
	{
		cuda_Filter_float << <threadsPerBlock, blocksPerGrid >> > ((float*)d_SrcData, d_Filter_Core, d_Result, stage,
			rootFilterInfo->CoreLength, SrcLen, *cFilter->scale);
	}
	break;
	}
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}
	
	CData* target_Data = (CData*)TargetData;
	h_Result = (FILTTED_DATA_TYPE*)target_Data->Buff + target_Data->Pos;
	err = hipMemcpy(h_Result, d_Result, result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_Result failed!\r\n");
	}
	UINT T;
	T = SrcData->ProcessPos;
	SrcData->ProcessPos = (T + (SrcLen >> 2)) & SrcData->Mask;
	T = target_Data->Pos;
	target_Data->Pos = (T + (SrcLen >> 2)) & target_Data->Mask;

	stage++;
	stage &= 0x3;
	//printf("Cuda Filtting Done.\r\n");

	ReleaseMutex(cFilter->hCoreMutex);
}