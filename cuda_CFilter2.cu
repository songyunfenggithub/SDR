#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "Debug.h"
#include "CData.h"
#include "CFilter.h"
#include "CDemodulatorAM.h"

#include "cuda_CFilter2.cuh"

using namespace METHOD;

cuda_CFilter2 clscudaMainFilter2;
cuda_CFilter2 clscudaMainFilter2Q;
cuda_CFilter2 clscudaAudioFilter2;

__global__ void
cuda_core_Filter2(
	const ADC_DATA_TYPE* src, unsigned int srclen,
	int stage,
	FILTTED_DATA_TYPE* decimation_cache, unsigned int decimation_factor_bit,
	const CFilter::FILTER_CORE_DATA_TYPE* core, unsigned int corelen,
	CFilter::FILTER_CORE_DATA_TYPE* filtted_result,
	float scale
)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cache_length_mask = (srclen >> decimation_factor_bit) - 1;
	unsigned int cache_stage_length = (srclen>> 2) >> decimation_factor_bit;
	unsigned int cache_stage_step = stage * cache_stage_length;
	unsigned int src_stage_step = stage * (srclen >> 2);
	unsigned int index = blockIdx.x;
	decimation_cache[cache_stage_step + index] = src[src_stage_step + (index << decimation_factor_bit)];
	__syncthreads();
	unsigned int i_stage_step = cache_stage_step + index - corelen;
	float d = 0;
	for (unsigned int i = 0; i < corelen; i++) {
		d += decimation_cache[(i_stage_step + i) & cache_length_mask] * core[i];
	}
	filtted_result[cache_stage_step + index] = d * scale;
}

__global__ void
cuda_core_Filter2_Demodulator(CFilter::FILTER_CORE_DATA_TYPE* filtted_result, FILTTED_DATA_TYPE* demodulator_result,
	int stage, int src_stage_length, unsigned int decimation_factor_bit, unsigned int cache_length)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cache_stage_length = src_stage_length >> decimation_factor_bit;
	unsigned int cache_stage_step = stage * cache_stage_length;

	unsigned int index = blockIdx.x;

	unsigned int n = 1 << 2;
	unsigned int mask = n - 1;
	unsigned int demodulator_result_index;
	float d;
	if ((index & mask) == 0) {
		demodulator_result_index = index >> 2;
		demodulator_result[demodulator_result_index] = 0;
		for (int i = 0; i < n; i++) {
			if ((d = filtted_result[cache_stage_step + index + i]) > 0) demodulator_result[demodulator_result_index] += d;
		}
	}
}

cuda_CFilter2::cuda_CFilter2()
{

}

cuda_CFilter2::~cuda_CFilter2()
{

}

void cuda_getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);
	DbgMsg("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	DbgMsg("max thread num: %d\n", prop.maxThreadsPerBlock);
	DbgMsg("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void cuda_CFilter2::Init(CFilter* f)
{
	UnInit();

	cuda_getThreadNum();

	hipError_t err = hipSuccess;

	cFilter = f;
	rootFilterInfo = &f->rootFilterInfo1;
	SrcData = f->SrcData;
	TargetData = f->TargetData;
	SrcLen = f->FilterSrcLen;

	DbgMsg("Cuda_Init [Vector addition of %d CUDA_FILTER_BUFF_SRC_LENGTH]\n", SrcLen);

	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_SrcData failed!\r\n");
			//goto Error;
		}
	}
	size_t src_data_size = SrcLen * sizeof(ADC_DATA_TYPE);
	err = hipMalloc((void**)&d_SrcData, src_data_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_SrcData failed!\r\n");
		//goto Error;
	}

	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filter_Core failed!\r\n");
			//goto Error;
		}
	}
	size_t filter_core_size = rootFilterInfo->CoreLength * sizeof(CFilter::FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipMemcpy(d_Filter_Core, rootFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		DbgMsg("hipMemcpy d_Filter_Core failed!\r\n");
		//goto Error;
	}

	if (d_Decimation_Cache != NULL) {
		err = hipFree(d_Decimation_Cache);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Decimation_Cache failed!\r\n");
			//goto Error;
		}
	}

	size_t decimation_cache_size = (SrcLen >> rootFilterInfo->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Decimation_Cache, decimation_cache_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	if (d_Filtted_Result != NULL) {
		err = hipFree(d_Filtted_Result);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filtted_Result failed!\r\n");
			//goto Error;
		}
	}
	size_t filter_result_size = (SrcLen >> rootFilterInfo->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Filtted_Result, filter_result_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Filtted_Result failed!\r\n");
		//goto Error;
	}

	if (d_Demodulator_Result != NULL) {
		err = hipFree(d_Demodulator_Result);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Demodulator_Result failed!\r\n");
			//goto Error;
		}
	}
	size_t demodulator_result_size = ((SrcLen >> 2) >> rootFilterInfo->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Demodulator_Result, demodulator_result_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	DbgMsg("Cuda Filter2 Init Done.\r\n");
}

void cuda_CFilter2::UnInit(void)
{
	if (this == NULL)return;
	hipError_t err = hipSuccess;

	// Free device global memory
	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_SrcData failed!\r\n");
			//goto Error;
		}
		d_SrcData = NULL;
	}
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filter_Core failed!\r\n");
			//goto Error;
		}
		d_Filter_Core = NULL;
	}
	if (d_Filtted_Result != NULL) {
		err = hipFree(d_Filtted_Result);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filtted_Result failed!\r\n");
			//goto Error;
		}
		d_Filtted_Result = NULL;
	}

	if (d_Decimation_Cache != NULL) {
		err = hipFree(d_Decimation_Cache);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Decimation_Cache failed!\r\n");
			//goto Error;
		}
		d_Decimation_Cache = NULL;
	}
	if (d_Demodulator_Result != NULL) {
		err = hipFree(d_Demodulator_Result);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Demodulator_Result failed!\r\n");
			//goto Error;
		}
		d_Demodulator_Result = NULL;
	}
	// Free host memory
	//free(h_Filter_Core);

	DbgMsg("cuda_CFilter2 UnInit Closed.\r\n");
}

void cuda_CFilter2::Filtting(void)
{
	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = (SrcLen >> 2) >> rootFilterInfo->decimationFactorBit;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;

	CData* src_Data = (CData*)SrcData;
	h_SrcData = (ADC_DATA_TYPE*)src_Data->Buff + src_Data->ProcessPos;
	//DbgMsg("stage:%d\r\n", stage);
	static size_t stage_size = (SrcLen >> 2) * sizeof(ADC_DATA_TYPE);
	err = hipMemcpy((char*)d_SrcData + stage * stage_size, h_SrcData, stage_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		DbgMsg("hipMemcpy d_SrcData failed!\r\n");
	}

	//DbgMsg("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		//const ADC_DATA_TYPE* src, unsigned int srclen,
		//int stage,
		//FILTTED_DATA_TYPE* decimation_cache, unsigned int decimation_factor_bit,
		//const CFilter::FILTER_CORE_DATA_TYPE* core, unsigned int corelen,
		//CFilter::FILTER_CORE_DATA_TYPE* filtted_result,
		//float scale

	cuda_core_Filter2 <<< threadsPerBlock, blocksPerGrid >>> (
		d_SrcData, SrcLen,
		stage,
		d_Decimation_Cache, rootFilterInfo->decimationFactorBit,
		d_Filter_Core, rootFilterInfo->CoreLength,
		d_Filtted_Result,
		*cFilter->scale
		);

	err = hipGetLastError();
	if (err != hipSuccess) {
		DbgMsg("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		DbgMsg("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}

	CData* target_Data = (CData*)TargetData;
	h_Filtted_Result = (FILTTED_DATA_TYPE*)target_Data->Buff + target_Data->Pos;
	err = hipMemcpy(h_Filtted_Result, 
		d_Filtted_Result + stage * ((SrcLen >> 2) >> rootFilterInfo->decimationFactorBit),
		((SrcLen >> 2) >> rootFilterInfo->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE),
		hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		DbgMsg("hipMemcpy h_Filtted_Result failed!\r\n");
	}
	UINT T;
	T = src_Data->ProcessPos;
	src_Data->ProcessPos = (T + (SrcLen >> 2)) & src_Data->Mask;
	T = target_Data->Pos;
	target_Data->Pos = (T + ((SrcLen >> 2) >> rootFilterInfo->decimationFactorBit)) & target_Data->Mask;

	/*
	h_Demodulator_Result = (FILTTED_DATA_TYPE*)clsDemodulatorAm.AMBuff + clsDemodulatorAm.AMPos;
	err = hipMemcpy(h_Demodulator_Result, d_Demodulator_Result, cuda_demodulator_result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		DbgMsg("hipMemcpy h_Demodulator_Result failed!\r\n");
	}
	clsDemodulatorAm.AMPos += cuda_demodulator_result_size / sizeof(FILTTED_DATA_TYPE);
	clsDemodulatorAm.AMPos &= DEMODULATOR_BUFF_LENGTH_MASK;
	*/

	stage++;
	stage &= 0x3;
}