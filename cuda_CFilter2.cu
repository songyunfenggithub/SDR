#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "myDebug.h"
#include "CData.h"
#include "CWaveFilter.h"
#include "CDemodulatorAM.h"

#include "cuda_CFilter2.cuh"

#define DEMODULATOR_FILTER_SAMPLERATE_OFFSET_BIT	0x3

cuda_CFilter2 clscudaFilter2;

__global__ void
cuda_core_Filter2(const ADCDATATYPE* src, FILTEDDATATYPE* decimation_cache, const FILTER_CORE_DATA_TYPE* core, FILTER_CORE_DATA_TYPE* filtted_result,
	int stage, unsigned int decimation_factor_bit, unsigned int corelen)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cache_length_mask = (CUDA_FILTER_BUFF_SRC_LENGTH >> decimation_factor_bit) - 1;
	unsigned int cache_stage_length = CUDA_FILTER_BUFF_STEP_LENGTH >> decimation_factor_bit;
	unsigned int cache_stage_step = stage * cache_stage_length;
	unsigned int src_stage_step = stage * CUDA_FILTER_BUFF_STEP_LENGTH;
	unsigned int index = blockIdx.x;
	
	decimation_cache[cache_stage_step + index] = src[src_stage_step + (index << decimation_factor_bit)];

	//���߳̿��е��߳̽���ͬ��
	__syncthreads();

	unsigned int i_stage_step = cache_stage_step + index - corelen;
	filtted_result[cache_stage_step + index] = 0;
	for (unsigned int i = 0; i < corelen; i++) {
		filtted_result[cache_stage_step + index] += decimation_cache[(i_stage_step + i) & cache_length_mask] * core[i];
	}
}

__global__ void
cuda_core_Filter2_Demodulator(FILTER_CORE_DATA_TYPE* filtted_result, FILTEDDATATYPE* demodulator_result,
	int stage, int src_stage_length, unsigned int decimation_factor_bit, unsigned int cache_length)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cache_stage_length = src_stage_length >> decimation_factor_bit;
	unsigned int cache_stage_step = stage * cache_stage_length;

	unsigned int index = blockIdx.x;

	unsigned int n = 1 << DEMODULATOR_FILTER_SAMPLERATE_OFFSET_BIT;
	unsigned int mask = n - 1;
	unsigned int demodulator_result_index;
	float d;
	if ((index & mask) == 0) {
		demodulator_result_index = index >> DEMODULATOR_FILTER_SAMPLERATE_OFFSET_BIT;
		demodulator_result[demodulator_result_index] = 0;
		for (int i = 0; i < n; i++) {
			if ((d = filtted_result[cache_stage_step + index + i]) > 0) demodulator_result[demodulator_result_index] += d;
		}
	}
}

cuda_CFilter2::cuda_CFilter2()
{

}

cuda_CFilter2::~cuda_CFilter2()
{

}

void cuda_getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);
	printf("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	printf("max thread num: %d\n", prop.maxThreadsPerBlock);
	printf("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void cuda_CFilter2::Init(CWaveFilter::PFILTERINFO pFilterInfo)
{
	UnInit();

	cuda_getThreadNum();

	hipError_t err = hipSuccess;

	rootFilterInfo = pFilterInfo;

	printf("Cuda_Init [Vector addition of %d CUDA_FILTER_BUFF_SRC_LENGTH]\n", CUDA_FILTER_BUFF_SRC_LENGTH);

	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			printf("hipFree d_SrcData failed!\r\n");
			//goto Error;
		}
	}
	size_t src_data_size = CUDA_FILTER_BUFF_SRC_LENGTH * sizeof(ADCDATATYPE);
	err = hipMalloc((void**)&d_SrcData, src_data_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_SrcData failed!\r\n");
		//goto Error;
	}

	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
			//goto Error;
		}
	}
	size_t filter_core_size = pFilterInfo->CoreLength * sizeof(FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipMemcpy(d_Filter_Core, pFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_Filter_Core failed!\r\n");
		//goto Error;
	}

	if (d_Decimation_Cache != NULL) {
		err = hipFree(d_Decimation_Cache);
		if (err != hipSuccess) {
			printf("hipFree d_Decimation_Cache failed!\r\n");
			//goto Error;
		}
	}

	size_t decimation_cache_size = (CUDA_FILTER_BUFF_SRC_LENGTH >> rootFilterInfo->decimationFactorBit) * sizeof(FILTEDDATATYPE);
	err = hipMalloc((void**)&d_Decimation_Cache, decimation_cache_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	if (d_Filtted_Result != NULL) {
		err = hipFree(d_Filtted_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Filtted_Result failed!\r\n");
			//goto Error;
		}
	}
	size_t filter_result_size = (CUDA_FILTER_BUFF_SRC_LENGTH >> rootFilterInfo->decimationFactorBit) * sizeof(FILTEDDATATYPE);
	err = hipMalloc((void**)&d_Filtted_Result, filter_result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Filtted_Result failed!\r\n");
		//goto Error;
	}

	if (d_Demodulator_Result != NULL) {
		err = hipFree(d_Demodulator_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Demodulator_Result failed!\r\n");
			//goto Error;
		}
	}
	size_t demodulator_result_size = (CUDA_FILTER_BUFF_STEP_LENGTH >> rootFilterInfo->decimationFactorBit) * sizeof(FILTEDDATATYPE);
	err = hipMalloc((void**)&d_Demodulator_Result, demodulator_result_size);
	if (err != hipSuccess) {
		printf("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	printf("Cuda Filter2 Init Done.\r\n");
}

void cuda_CFilter2::UnInit(void)
{
	hipError_t err = hipSuccess;

	// Free device global memory
	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			printf("hipFree d_SrcData failed!\r\n");
			//goto Error;
		}
		d_SrcData = NULL;
	}
	if (d_Filter_Core != NULL) {
		err = hipFree(d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree d_Filter_Core failed!\r\n");
			//goto Error;
		}
		d_Filter_Core = NULL;
	}
	if (d_Filtted_Result != NULL) {
		err = hipFree(d_Filtted_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Filtted_Result failed!\r\n");
			//goto Error;
		}
		d_Filtted_Result = NULL;
	}

	if (d_Decimation_Cache != NULL) {
		err = hipFree(d_Decimation_Cache);
		if (err != hipSuccess) {
			printf("hipFree d_Decimation_Cache failed!\r\n");
			//goto Error;
		}
		d_Decimation_Cache = NULL;
	}
	if (d_Demodulator_Result != NULL) {
		err = hipFree(d_Demodulator_Result);
		if (err != hipSuccess) {
			printf("hipFree d_Demodulator_Result failed!\r\n");
			//goto Error;
		}
		d_Demodulator_Result = NULL;
	}
	// Free host memory
	//free(h_Filter_Core);

	printf("cuda_CFilter2 UnInit Closed.\r\n");
}

void cuda_CFilter2::Filtting(void)
{
	WaitForSingleObject(clsWaveFilter.hCoreMutex, INFINITE);

	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = CUDA_FILTER_BUFF_STEP_LENGTH >> rootFilterInfo->decimationFactorBit;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;

	h_SrcData = (ADCDATATYPE*)clsData.AdcBuff + clsData.FilttedPos;
	//printf("stage:%d\r\n", stage);
	static size_t stage_size = CUDA_FILTER_BUFF_STEP_LENGTH * sizeof(ADCDATATYPE);
	err = hipMemcpy((char*)d_SrcData + stage * stage_size, h_SrcData, stage_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy d_SrcData failed!\r\n");
	}

	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	cuda_core_Filter2 << <threadsPerBlock, blocksPerGrid >> > (d_SrcData, d_Decimation_Cache, d_Filter_Core, d_Filtted_Result,
		stage, rootFilterInfo->decimationFactorBit, rootFilterInfo->CoreLength);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}

	h_Filtted_Result = (FILTEDDATATYPE*)clsData.FilttedBuff + clsData.FilttedBuffPos;
	err = hipMemcpy(h_Filtted_Result, 
		d_Filtted_Result + stage * (CUDA_FILTER_BUFF_STEP_LENGTH >> rootFilterInfo->decimationFactorBit), 
		(CUDA_FILTER_BUFF_STEP_LENGTH >> rootFilterInfo->decimationFactorBit) * sizeof(FILTEDDATATYPE), 
		hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_Filtted_Result failed!\r\n");
	}
	UINT T;
	T = clsData.FilttedPos;
	clsData.FilttedPos = (T + CUDA_FILTER_BUFF_STEP_LENGTH) & DATA_BUFFER_MASK;
	T = clsData.FilttedBuffPos;
	clsData.FilttedBuffPos = (T + (CUDA_FILTER_BUFF_STEP_LENGTH >> rootFilterInfo->decimationFactorBit)) & DATA_BUFFER_MASK;

	/*
	h_Demodulator_Result = (FILTEDDATATYPE*)clsDemodulatorAm.AMBuff + clsDemodulatorAm.AMPos;
	err = hipMemcpy(h_Demodulator_Result, d_Demodulator_Result, cuda_demodulator_result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy h_Demodulator_Result failed!\r\n");
	}
	clsDemodulatorAm.AMPos += cuda_demodulator_result_size / sizeof(FILTEDDATATYPE);
	clsDemodulatorAm.AMPos &= DEMODULATOR_BUFF_LENGTH_MASK;
	*/

	stage++;
	stage &= 0x3;

	ReleaseMutex(clsWaveFilter.hCoreMutex);
}