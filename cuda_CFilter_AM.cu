#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "Debug.h"
#include "CData.h"
#include "CFilter.h"
#include "CAM.h"

#include "cuda_CFilter_AM.cuh"

using namespace METHOD;

cuda_CFilter_AM clscudaMainFilter_AM;

__global__ void
cuda_core_Filter_AM(
	const ADC_DATA_TYPE* src, int stage, unsigned int srclen,
	FILTTED_DATA_TYPE* decimation_cache1, const CFilter::FILTER_CORE_DATA_TYPE* core1, unsigned int decimation_factor_bit1, unsigned int corelen1,
	FILTTED_DATA_TYPE* decimation_cache2, const CFilter::FILTER_CORE_DATA_TYPE* core2, unsigned int decimation_factor_bit2, unsigned int corelen2,
	CFilter::FILTER_CORE_DATA_TYPE* filtted_result1,
	CFilter::FILTER_CORE_DATA_TYPE* filtted_result2,
	float scale
)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cache1_length_mask = (srclen >> decimation_factor_bit1) - 1;
	unsigned int cache1_stage_length = (srclen>> 2) >> decimation_factor_bit1;
	unsigned int cache1_stage_step = stage * cache1_stage_length;
	unsigned int src_stage_step = stage * (srclen >> 2);
	unsigned int index = blockIdx.x;
	decimation_cache1[cache1_stage_step + index] = src[src_stage_step + (index << decimation_factor_bit1)];
	__syncthreads();
	unsigned int index_mask = (1 << decimation_factor_bit2) - 1;
	unsigned int i_stage_step = cache1_stage_step + index - corelen1;
	float d = 0;
	if (!(index & index_mask)) 
	{
		for (unsigned int i = 0; i < corelen1; i++) {
			d += decimation_cache1[(i_stage_step + i) & cache1_length_mask] * core1[i];
		}
		filtted_result1[cache1_stage_step + index] = d;
	}
	__syncthreads();
	unsigned int cache2_length_mask = (srclen >> (decimation_factor_bit1 + decimation_factor_bit2)) - 1;
	unsigned int cache2_stage_length = (srclen >> 2) >> (decimation_factor_bit1 + decimation_factor_bit2);
	unsigned int cache2_stage_step = stage * cache2_stage_length;
	unsigned int index2 = index >> decimation_factor_bit2;
	if(!(index & index_mask))
		decimation_cache2[cache2_stage_step + index2] = filtted_result1[cache1_stage_step + index];
	__syncthreads();
	i_stage_step = cache2_stage_step + index2 - corelen2;
	d = 0;
	for (unsigned int i = 0; i < corelen2; i++) {
		d += decimation_cache2[(i_stage_step + i) & cache2_length_mask] * core2[i];
	}
	filtted_result2[cache2_stage_step + index2] = d * scale;
}

__global__ void
cuda_core_Filter_AM_Demodulator(CFilter::FILTER_CORE_DATA_TYPE* filtted_result, FILTTED_DATA_TYPE* demodulator_result,
	int stage, int src_stage_length, unsigned int decimation_factor_bit, unsigned int cache_length)
{
	//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int cache_stage_length = src_stage_length >> decimation_factor_bit;
	unsigned int cache_stage_step = stage * cache_stage_length;

	unsigned int index = blockIdx.x;

	unsigned int n = 1 << 2;
	unsigned int mask = n - 1;
	unsigned int demodulator_result_index;
	float d;
	if ((index & mask) == 0) {
		demodulator_result_index = index >> 2;
		demodulator_result[demodulator_result_index] = 0;
		for (int i = 0; i < n; i++) {
			if ((d = filtted_result[cache_stage_step + index + i]) > 0) demodulator_result[demodulator_result_index] += d;
		}
	}
}

cuda_CFilter_AM::cuda_CFilter_AM()
{

}

cuda_CFilter_AM::~cuda_CFilter_AM()
{

}

void cuda_CFilter_AM::Init(CFilter* f)
{
	hipError_t err = hipSuccess;

	cFilter = f;
	rootFilterInfo1 = &f->rootFilterInfo1;
	rootFilterInfo2 = &f->rootFilterInfo2;
	SrcData = f->SrcData;
	TargetData = f->TargetData;
	SrcLen = f->FilterSrcLen;

	DbgMsg("Cuda_Init [Vector addition of %d CUDA_FILTER_BUFF_SRC_LENGTH]\n", SrcLen);

	UnInit();

	size_t src_data_size = SrcLen * sizeof(ADC_DATA_TYPE);
	err = hipMalloc((void**)&d_SrcData, src_data_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_SrcData failed!\r\n");
		//goto Error;
	}

	size_t filter_core_size1 = rootFilterInfo1->CoreLength * sizeof(CFilter::FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core1, filter_core_size1);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipMemcpy(d_Filter_Core1, rootFilterInfo1->FilterCore, filter_core_size1, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		DbgMsg("hipMemcpy d_Filter_Core failed!\r\n");
		//goto Error;
	}
	size_t filter_core_size2 = rootFilterInfo2->CoreLength * sizeof(CFilter::FILTER_CORE_DATA_TYPE);
	err = hipMalloc((void**)&d_Filter_Core2, filter_core_size2);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipMemcpy(d_Filter_Core2, rootFilterInfo2->FilterCore, filter_core_size2, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		DbgMsg("hipMemcpy d_Filter_Core failed!\r\n");
		//goto Error;
	}

	size_t decimation_cache_size1 = (SrcLen >> rootFilterInfo1->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Decimation_Cache1, decimation_cache_size1);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}
	size_t decimation_cache_size2 = (SrcLen >> (rootFilterInfo1->decimationFactorBit + rootFilterInfo2->decimationFactorBit)) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Decimation_Cache2, decimation_cache_size2);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}

	size_t filter_result_size1 = (SrcLen >> rootFilterInfo1->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Filtted_Result1, filter_result_size1);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Filtted_Result failed!\r\n");
		//goto Error;
	}
	size_t filter_result_size2 = (SrcLen >> (rootFilterInfo1->decimationFactorBit + rootFilterInfo2->decimationFactorBit)) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Filtted_Result2, filter_result_size2);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Filtted_Result failed!\r\n");
		//goto Error;
	}

	/*
	size_t demodulator_result_size = ((srcLen >> 2) >> rootFilterInfo->decimationFactorBit) * sizeof(FILTTED_DATA_TYPE);
	err = hipMalloc((void**)&d_Demodulator_Result, demodulator_result_size);
	if (err != hipSuccess) {
		DbgMsg("hipMalloc d_Demodulator_Result failed!\r\n");
		//goto Error;
	}
	*/
	DbgMsg("Cuda Filter3 Init.\r\n");
}

void cuda_CFilter_AM::UnInit(void)
{
	if (this == NULL)return;
	hipError_t err = hipSuccess;

	// Free device global memory
	if (d_SrcData != NULL) {
		err = hipFree(d_SrcData);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_SrcData failed!\r\n");
			//goto Error;
		}
		d_SrcData = NULL;
	}
	if (d_Filter_Core1 != NULL) {
		err = hipFree(d_Filter_Core1);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filter_Core1 failed!\r\n");
			//goto Error;
		}
		d_Filter_Core1 = NULL;
	}
	if (d_Filter_Core2 != NULL) {
		err = hipFree(d_Filter_Core2);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filter_Core2 failed!\r\n");
			//goto Error;
		}
		d_Filter_Core2 = NULL;
	}
	if (d_Filtted_Result1 != NULL) {
		err = hipFree(d_Filtted_Result1);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filtted_Result1 failed!\r\n");
			//goto Error;
		}
		d_Filtted_Result1 = NULL;
	}
	if (d_Filtted_Result2 != NULL) {
		err = hipFree(d_Filtted_Result2);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Filtted_Result failed!\r\n");
			//goto Error;
		}
		d_Filtted_Result2 = NULL;
	}

	if (d_Decimation_Cache1 != NULL) {
		err = hipFree(d_Decimation_Cache1);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Decimation_Cache1 failed!\r\n");
			//goto Error;
		}
		d_Decimation_Cache1 = NULL;
	}
	if (d_Decimation_Cache2 != NULL) {
		err = hipFree(d_Decimation_Cache2);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Decimation_Cache2 failed!\r\n");
			//goto Error;
		}
		d_Decimation_Cache2 = NULL;
	}
	if (d_Demodulator_Result != NULL) {
		err = hipFree(d_Demodulator_Result);
		if (err != hipSuccess) {
			DbgMsg("hipFree d_Demodulator_Result failed!\r\n");
			//goto Error;
		}
		d_Demodulator_Result = NULL;
	}
	DbgMsg("cuda_CFilter_AM UnInit.\r\n");
}

void cuda_CFilter_AM::Filtting(void)
{
	WaitForSingleObject(cFilter->hCoreMutex, INFINITE);

	if (cFilter->Cuda_Filter_N_New != cFilter->Cuda_Filter_N_Doing) {
		ReleaseMutex(cFilter->hCoreMutex);
		Sleep(100);
		return;
	}

	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = SrcLen >> (2 + rootFilterInfo1->decimationFactorBit);
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;

	h_SrcData = (ADC_DATA_TYPE*)SrcData->Buff + SrcData->ProcessPos;
	//DbgMsg("stage:%d\r\n", stage);
	size_t stage_size = (SrcLen >> 2) * sizeof(ADC_DATA_TYPE);
	err = hipMemcpy((char*)d_SrcData + stage * stage_size, h_SrcData, stage_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		DbgMsg("3cudaMemcpy d_SrcData failed!\r\n");
	}

	//DbgMsg("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		//const ADC_DATA_TYPE* src, int stage, unsigned int srclen,
		//FILTTED_DATA_TYPE* decimation_cache1, const CFilter::FILTER_CORE_DATA_TYPE* core1, unsigned int decimation_factor_bit1, unsigned int corelen1,
		//FILTTED_DATA_TYPE* decimation_cache2, const CFilter::FILTER_CORE_DATA_TYPE* core2, unsigned int decimation_factor_bit2, unsigned int corelen2,
		//CFilter::FILTER_CORE_DATA_TYPE* filtted1_result,
		//CFilter::FILTER_CORE_DATA_TYPE* filtted2_result

	cuda_core_Filter_AM << <threadsPerBlock, blocksPerGrid >> > (
		d_SrcData, stage, SrcLen,
		d_Decimation_Cache1, d_Filter_Core1, rootFilterInfo1->decimationFactorBit, rootFilterInfo1->CoreLength,
		d_Decimation_Cache2, d_Filter_Core2, rootFilterInfo2->decimationFactorBit, rootFilterInfo2->CoreLength,
		d_Filtted_Result1,
		d_Filtted_Result2,
		*cFilter->Scale
		);
	err = hipGetLastError();
	if (err != hipSuccess) {
		DbgMsg("3cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		DbgMsg("3cudaDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}

	CData* target_Data = (CData*)TargetData;
	h_Filtted_Result2 = (FILTTED_DATA_TYPE*)target_Data->Buff + target_Data->Pos;
	size_t result_size = (SrcLen >> 2) >> (rootFilterInfo1->decimationFactorBit + rootFilterInfo2->decimationFactorBit);
	err = hipMemcpy(
		h_Filtted_Result2, 
		d_Filtted_Result2 + stage * result_size, 
		result_size * sizeof(FILTTED_DATA_TYPE),
		hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		DbgMsg("3cudaMemcpy h_Filtted_Result failed!\r\n");
	}
	UINT T;
	T = SrcData->ProcessPos;
	SrcData->ProcessPos = (T + (SrcLen >> 2)) & SrcData->Mask;
	T = target_Data->Pos;
	target_Data->Pos = (T + result_size) & target_Data->Mask;

	stage++;
	stage &= 0x3;

	ReleaseMutex(cFilter->hCoreMutex);
}