#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <stdlib.h>
#include <time.h>
#include <Windows.h>

#include "public.h"
#include "CData.h"
#include "myDebug.h"
#include "CWaveFFT.h"
#include "CFilter.h"

#include "cuda_FFT.cuh"

//using namespace WINS;
//using namespace DEVICES;

#define CUFFT_CALL( call )                                                                                             \
    {                                                                                                                  \
        auto status = static_cast<hipfftResult>( call );                                                                \
        if ( status != HIPFFT_SUCCESS )                                                                                 \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUFFT call \"%s\" in line %d of file %s failed "                                          \
                     "with "                                                                                           \
                     "code (%d).\n",                                                                                   \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     status );                                                                                         \
    }

hipfftDoubleComplex* cuda_FFT_CompiData = NULL;
hipfftDoubleComplex* cuda_FFT_CompoData = NULL;
hipfftDoubleComplex* cuda_FFT_d_fftData = NULL;
hipfftDoubleComplex* cuda_FFT_d_outfftData = NULL;
hipfftHandle cuda_FFT_fft_plan = 0;// cuda library function handle

hipfftDoubleComplex* cuda_FFT_CompiData_filtted = NULL;
hipfftDoubleComplex* cuda_FFT_CompoData_filtted = NULL;
hipfftDoubleComplex* cuda_FFT_d_fftData_filtted = NULL;
hipfftDoubleComplex* cuda_FFT_d_outfftData_filtted = NULL;
hipfftHandle cuda_FFT_fft_plan_filtted = 0;// cuda library function handle

void cuda_FFT(WHICHSIGNAL WhichSignal)
{
    float cost, s;
    s = GetTickCount();
    if (WhichSignal == WHICHSIGNAL::SIGNAL_ORIGNAL) {
        CUFFT_CALL(hipMemcpy(cuda_FFT_d_fftData, cuda_FFT_CompiData, clsWaveFFT.FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));// copy data from host to device
        //WaitForSingleObject(cuda_FFT_hMutexBuff, INFINITE);
        CUFFT_CALL(hipfftExecZ2Z(cuda_FFT_fft_plan, (hipfftDoubleComplex*)cuda_FFT_d_fftData, (hipfftDoubleComplex*)cuda_FFT_d_outfftData, HIPFFT_FORWARD));//execute
        CUFFT_CALL(hipDeviceSynchronize());//wait to be done
        //ReleaseMutex(cuda_FFT_hMutexBuff);
        CUFFT_CALL(hipMemcpy(cuda_FFT_CompoData, cuda_FFT_d_outfftData, clsWaveFFT.FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));// copy the result from device to host
        //printf("Time of cudaFFT: %fms\r\n", GetTickCount() - s);
    }
    else {
        UINT n = (clsWaveFFT.FFTSize >> clsFilter.rootFilterInfo.decimationFactorBit);
        CUFFT_CALL(hipMemcpy(cuda_FFT_d_fftData_filtted, cuda_FFT_CompiData_filtted, n * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));// copy data from host to device
        //WaitForSingleObject(cuda_FFT_hMutexBuff, INFINITE);
        CUFFT_CALL(hipfftExecZ2Z(cuda_FFT_fft_plan_filtted, (hipfftDoubleComplex*)cuda_FFT_d_fftData_filtted, (hipfftDoubleComplex*)cuda_FFT_d_outfftData_filtted, HIPFFT_FORWARD));//execute
        CUFFT_CALL(hipDeviceSynchronize());//wait to be done
        //ReleaseMutex(cuda_FFT_hMutexBuff);
        CUFFT_CALL(hipMemcpy(cuda_FFT_CompoData_filtted, cuda_FFT_d_outfftData_filtted, n * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));// copy the result from device to host
        //printf("Time of cudaFFT: %fms\r\n", GetTickCount() - s);
    }
}

void cuda_FFT_Init(void)
{
    UINT FFTSize = clsWaveFFT.FFTSize;

    if(cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    cuda_FFT_CompiData = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    cuda_FFT_CompoData = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host
      
    if(cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    hipMalloc((void**)&cuda_FFT_d_fftData, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    hipMalloc((void**)&cuda_FFT_d_outfftData, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if(cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);
    hipfftPlan1d(&cuda_FFT_fft_plan, FFTSize, HIPFFT_Z2Z, 1);//declaration

    FFTSize = clsWaveFFT.FFTSize >> clsFilter.rootFilterInfo.decimationFactorBit;
    if (cuda_FFT_CompiData_filtted != NULL) free(cuda_FFT_CompiData_filtted);
    cuda_FFT_CompiData_filtted = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_CompoData_filtted != NULL) free(cuda_FFT_CompoData_filtted);
    cuda_FFT_CompoData_filtted = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_d_fftData_filtted != NULL) hipFree(cuda_FFT_d_fftData_filtted);
    hipMalloc((void**)&cuda_FFT_d_fftData_filtted, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_d_outfftData_filtted != NULL)  hipFree(cuda_FFT_d_outfftData_filtted);
    hipMalloc((void**)&cuda_FFT_d_outfftData_filtted, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_fft_plan_filtted != 0) hipfftDestroy(cuda_FFT_fft_plan_filtted);
    hipfftPlan1d(&cuda_FFT_fft_plan_filtted, FFTSize >> clsFilter.rootFilterInfo.decimationFactorBit, HIPFFT_Z2Z, 1);//declaration
}

void cuda_FFT_UnInit(void)
{
    clsWaveFFT.FFTDoing = false;
    while(clsWaveFFT.FFTThreadExit == false);

    if (cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    if (cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    if (cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);

    if (cuda_FFT_CompiData_filtted != NULL) free(cuda_FFT_CompiData_filtted);
    if (cuda_FFT_CompoData_filtted != NULL) free(cuda_FFT_CompoData_filtted);
    if (cuda_FFT_d_fftData_filtted != NULL) hipFree(cuda_FFT_d_fftData_filtted);
    if (cuda_FFT_d_outfftData_filtted != NULL) hipFree(cuda_FFT_d_outfftData_filtted);
    if (cuda_FFT_fft_plan_filtted != 0) hipfftDestroy(cuda_FFT_fft_plan_filtted);
    DbgMsg("cuda_FFT_UnInit Closed.\r\n");
}

void cuda_FFT(WHICHSIGNAL WhichSignal, uint32_t pos)
{
    cuda_FFT_Prepare_Data(WhichSignal, pos);
    cuda_FFT(WhichSignal);
}

void cuda_FFT_Prepare_Data(WHICHSIGNAL WhichSignal, uint32_t pos)
{
    int i;
    if (WhichSignal == WHICHSIGNAL::SIGNAL_ORIGNAL) {
        memset(cuda_FFT_CompiData, 0, clsWaveFFT.FFTSize * sizeof(hipfftDoubleComplex));
        switch (AdcData->DataType) {
        case short_type:
        {
            short* Buff = (short*)AdcData->Buff;
            for (i = 0; i < clsWaveFFT.FFTStep; i++, pos++) {
                cuda_FFT_CompiData[i].x = (double)Buff[pos & AdcData->Mask];
                cuda_FFT_CompiData[i].y = 0;
            }
        }
        break;
        case float_type:
        {
            float* Buff = (float*)AdcData->Buff;
            for (i = 0; i < clsWaveFFT.FFTStep; i++, pos++) {
                cuda_FFT_CompiData[i].x = (double)Buff[pos & AdcData->Mask];
                cuda_FFT_CompiData[i].y = 0;
            }
        }
        break;
        }
    }
    else {
        UINT n = clsWaveFFT.FFTStep >> clsFilter.rootFilterInfo.decimationFactorBit;
        memset(cuda_FFT_CompiData_filtted, 0, n * sizeof(hipfftDoubleComplex));
        switch (AdcData->DataType) {
        case short_type:
        {
            short* Buff = (short*)AdcDataFiltted->Buff;
            for (i = 0; i < n; i++, pos++) {
                cuda_FFT_CompiData_filtted[i].x = (double)Buff[pos & AdcDataFiltted->Mask];
                cuda_FFT_CompiData_filtted[i].y = 0;
            }
        }
        break;
        case float_type:
        {
            float* Buff = (float*)AdcDataFiltted->Buff;
            for (i = 0; i < n; i++, pos++) {
                cuda_FFT_CompiData_filtted[i].x = (double)Buff[pos & AdcDataFiltted->Mask];
                cuda_FFT_CompiData_filtted[i].y = 0;
            }
        }
        break;
        }
    }
}

void cuda_FFT_Prepare_Data_for_MaxValue(WHICHSIGNAL WhichSignal, double *buff)
{
    UINT fftsize;
    if (WhichSignal == WHICHSIGNAL::SIGNAL_ORIGNAL) {
        fftsize = clsWaveFFT.FFTStep;
        for (int i = 0; i < fftsize; i++) {
            cuda_FFT_CompiData[i].x = (double)buff[i];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    else {
        fftsize = clsWaveFFT.FFTStep >> clsFilter.rootFilterInfo.decimationFactorBit;
        for (int i = 0; i < fftsize; i++) {
            cuda_FFT_CompiData_filtted[i].x = (double)buff[i];
            cuda_FFT_CompiData_filtted[i].y = 0;
        }
    }
}
