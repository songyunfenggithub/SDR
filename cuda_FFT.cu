#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#include <stdlib.h>
#include <time.h>
#include <Windows.h>
#include "CWaveData.h"
#include "CWaveFFT.h"

#include "cuda_FFT.cuh"


#define CUFFT_CALL( call )                                                                                             \
    {                                                                                                                  \
        auto status = static_cast<hipfftResult>( call );                                                                \
        if ( status != HIPFFT_SUCCESS )                                                                                 \
            fprintf( stderr,                                                                                           \
                     "ERROR: CUFFT call \"%s\" in line %d of file %s failed "                                          \
                     "with "                                                                                           \
                     "code (%d).\n",                                                                                   \
                     #call,                                                                                            \
                     __LINE__,                                                                                         \
                     __FILE__,                                                                                         \
                     status );                                                                                         \
    }

hipfftDoubleComplex* cuda_FFT_CompiData = NULL;
hipfftDoubleComplex* cuda_FFT_CompoData = NULL;
hipfftDoubleComplex* cuda_FFT_d_fftData = NULL;
hipfftDoubleComplex* cuda_FFT_d_outfftData = NULL;
hipfftHandle cuda_FFT_fft_plan = 0;// cuda library function handle

void cuda_FFT(void)
{
    float cost, s;
    s = GetTickCount();
    CUFFT_CALL(hipMemcpy(cuda_FFT_d_fftData, cuda_FFT_CompiData, clsWaveFFT.FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyHostToDevice));// copy data from host to device
    //WaitForSingleObject(cuda_FFT_hMutexBuff, INFINITE);
    CUFFT_CALL(hipfftExecZ2Z(cuda_FFT_fft_plan, (hipfftDoubleComplex*)cuda_FFT_d_fftData, (hipfftDoubleComplex*)cuda_FFT_d_outfftData, HIPFFT_FORWARD));//execute
    CUFFT_CALL(hipDeviceSynchronize());//wait to be done
    //ReleaseMutex(cuda_FFT_hMutexBuff);
    CUFFT_CALL(hipMemcpy(cuda_FFT_CompoData, cuda_FFT_d_outfftData, clsWaveFFT.FFTSize * sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost));// copy the result from device to host
    //printf("Time of cudaFFT: %fms\r\n", GetTickCount() - s);
}

void cuda_FFT_Init(void)
{
    UINT FFTSize = clsWaveFFT.FFTSize;
    if(cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    cuda_FFT_CompiData = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host

    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    cuda_FFT_CompoData = (hipfftDoubleComplex*)malloc(FFTSize * sizeof(hipfftDoubleComplex));//allocate memory for the data in host
      
    if(cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    hipMalloc((void**)&cuda_FFT_d_fftData, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    hipMalloc((void**)&cuda_FFT_d_outfftData, FFTSize * sizeof(hipfftDoubleComplex));// allocate memory for the data in device

    if(cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);
    hipfftPlan1d(&cuda_FFT_fft_plan, FFTSize, HIPFFT_Z2Z, 1);//declaration
}

void cuda_FFT_UnInit(void)
{
    while(clsWaveFFT.FFTThreadExit == false);

    if (cuda_FFT_CompiData != NULL) free(cuda_FFT_CompiData);
    if (cuda_FFT_CompoData != NULL) free(cuda_FFT_CompoData);
    if (cuda_FFT_d_fftData != NULL)  hipFree(cuda_FFT_d_fftData);
    if (cuda_FFT_d_outfftData != NULL)  hipFree(cuda_FFT_d_outfftData);
    if (cuda_FFT_fft_plan != 0) hipfftDestroy(cuda_FFT_fft_plan);
    printf("Cuda_FFT_Closed.\r\n");
}

void cuda_FFT(WHICHSIGNAL WhichSignal, uint32_t pos)
{
    cuda_FFT_Prepare_Data(WhichSignal, pos);
    cuda_FFT();
}

void cuda_FFT_Prepare_Data(WHICHSIGNAL WhichSignal, uint32_t pos)
{
    int i;
    memset(cuda_FFT_CompiData, 0, clsWaveFFT.FFTSize * sizeof(hipfftDoubleComplex));
    if (WhichSignal == WHICHSIGNAL::SIGNAL_ORIGNAL) {
        for (i = 0; i < clsWaveFFT.FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = (double)clsWaveData.AdcBuff[pos & DATA_BUFFER_MASK];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
    else {
        for (i = 0; i < clsWaveFFT.FFTStep; i++, pos++) {
            cuda_FFT_CompiData[i].x = (double)clsWaveData.FilttedBuff[pos & DATA_BUFFER_MASK];
            cuda_FFT_CompiData[i].y = 0;
        }
    }
}

void cuda_FFT_Prepare_Data_for_MaxValue(double *buff)
{
    for (int i = 0; i < clsWaveFFT.FFTSize; i++) {
        cuda_FFT_CompiData[i].x = (double)buff[i];
        cuda_FFT_CompiData[i].y = 0;
    }
}
