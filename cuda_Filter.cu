#include "hip/hip_runtime.h"

#include "stdafx.h"

#include <stdio.h>
#include <random>

#include <hip/hip_runtime.h>
#include <>

#include "CWaveData.h"
#include "CWaveFilter.h"

#include "cuda_Filter.cuh"


FILTERCOREDATATYPE* cuda_Filter_d_Filter_Core = NULL;
ADCDATATYPE* cuda_Filter_d_SrcData = NULL;
FILTEDDATATYPE* cuda_Filter_d_Result = NULL;

//FILTERCOREDATATYPE* h_Filter_Core = NULL;
ADCDATATYPE* cuda_Filter_h_SrcData = NULL;
FILTEDDATATYPE* cuda_Filter_h_Result = NULL;

size_t cuda_Filter_thread_size;
size_t cuda_Filter_result_size;
size_t cuda_Filter_src_step_size;

CWaveFilter::PFILTERINFO cuda_Filter_rootFilterInfo = NULL;

__global__ void
cuda_Filter(const ADCDATATYPE* src, const FILTERCOREDATATYPE* core, FILTEDDATATYPE* result, int stage, unsigned int corelen)
{
//	int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int index = blockIdx.x;
	unsigned int step = stage * CUDA_FILTER_BUFF_STEP_LENGTH;
	unsigned int istep = step + index - corelen;
	result[index] = 0;
	for (int n = 0; n < corelen; n++) {
		result[index] += src[(istep + n) & CUDA_FILTER_BUFF_SRC_LENGTH_MASK] * core[n];
	}
}

void Cuda_ReInitFilterCore(CWaveFilter::PFILTERINFO pFilterInfo)
{
	hipError_t err = hipSuccess;

	cuda_Filter_rootFilterInfo = pFilterInfo;

	if (cuda_Filter_d_Filter_Core != NULL) {
		err = hipFree(cuda_Filter_d_Filter_Core);
		if (err != hipSuccess) {
			printf("hipFree cuda_Filter_d_Filter_Core failed!\r\n");
			//goto Error;
		}
	}
	//if(h_Filter_Core != NULL)free(h_Filter_Core);

	size_t filter_core_size = pFilterInfo->CoreLength * sizeof(FILTERCOREDATATYPE);
	// Allocate the host input vector Core
	//h_Filter_Core = (FILTERCOREDATATYPE*)malloc(filter_core_size);

	//memcpy(h_Filter_Core, clsWaveFilter.FilterCore, filter_core_size);

	// Allocate the device input vector Filter_Core
	err = hipMalloc((void**)&cuda_Filter_d_Filter_Core, filter_core_size);
	if (err != hipSuccess) {
		printf("hipMalloc cuda_Filter_d_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipMemcpy(cuda_Filter_d_Filter_Core, pFilterInfo->FilterCore, filter_core_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy cuda_Filter_d_Filter_Core failed!\r\n");
		//goto Error;
	}

	printf("Cuda ReInit Filter Core Done.\r\n");

}

void cuda_getThreadNum(void)
{
	hipDeviceProp_t prop;
	int count;

	hipGetDeviceCount(&count);
	printf("gpu num %d\n", count);
	hipGetDeviceProperties(&prop, 0);
	printf("max thread num: %d\n", prop.maxThreadsPerBlock);
	printf("max grid dimensions: %d, %d, %d)\n",
		prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

void Cuda_Filter_Init(void)
{
	
	cuda_getThreadNum();

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;

	// Print the vector length to be used, and compute its size
	int numElements = CUDA_FILTER_BUFF_SRC_LENGTH;
	size_t src_data_size = numElements * sizeof(ADCDATATYPE);
	cuda_Filter_src_step_size = CUDA_FILTER_BUFF_STEP_LENGTH * sizeof(ADCDATATYPE);
	printf("Cuda_Init [Vector addition of %d elements]\n", numElements);

	// Allocate the host input vector A
	//cuda_Filter_h_SrcData = (ADCDATATYPE*)malloc(src_data_size);
	
	cuda_Filter_thread_size = CUDA_FILTER_BUFF_STEP_LENGTH;
	cuda_Filter_result_size = cuda_Filter_thread_size * sizeof(FILTEDDATATYPE);
	// Allocate the host output vector Result
	//float* cuda_Filter_h_Result = (float*)malloc(cuda_Filter_result_size);

	// Allocate the device input vector A
	err = hipMalloc((void**)&cuda_Filter_d_SrcData, src_data_size);
	if (err != hipSuccess) {
		printf("hipMalloc cuda_Filter_d_SrcData failed!\r\n");
		//goto Error;
	}
	// Allocate the device output vector C
	err = hipMalloc((void**)&cuda_Filter_d_Result, cuda_Filter_result_size);
	if (err != hipSuccess) {
		printf("hipMalloc cuda_Filter_d_Result failed!\r\n");
		//goto Error;
	}
	
	printf("Cuda Init Done.\r\n");
}

void Cuda_Filter_UnInit(void)
{

	while (clsWaveFilter.cud_Filter_exit == false);

	hipError_t err = hipSuccess;
	
	// Free device global memory
	err = hipFree(cuda_Filter_d_SrcData);
	if (err != hipSuccess) {
		printf("hipFree cuda_Filter_d_SrcData failed!\r\n");
		//goto Error;
	}
	err = hipFree(cuda_Filter_d_Filter_Core);
	if (err != hipSuccess) {
		printf("hipFree cuda_Filter_d_Filter_Core failed!\r\n");
		//goto Error;
	}
	err = hipFree(cuda_Filter_d_Result);
	if (err != hipSuccess) {
		printf("hipFree cuda_Filter_d_Result failed!\r\n");
		//goto Error;
	}

	// Free host memory
	//free(h_Filter_Core);

	printf("Cuda_Filter_Closed.\r\n");
}

void cuda_Filtting(void)
{
	hipError_t err = hipSuccess;
	static unsigned int stage = 0;

	size_t threadsPerBlock = cuda_Filter_thread_size;
	//int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
	size_t blocksPerGrid = 1;

	cuda_Filter_h_SrcData = (ADCDATATYPE*)clsWaveData.AdcBuff + clsWaveData.FilttedPos;
	//printf("stage:%d\r\n", stage);
	err = hipMemcpy((char*)cuda_Filter_d_SrcData + stage * cuda_Filter_src_step_size, cuda_Filter_h_SrcData, cuda_Filter_src_step_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy cuda_Filter_d_SrcData failed!\r\n");
	}

	WaitForSingleObject(clsWaveFilter.hCoreMutex, INFINITE);

	//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	cuda_Filter << <threadsPerBlock, blocksPerGrid >> > (cuda_Filter_d_SrcData, cuda_Filter_d_Filter_Core, cuda_Filter_d_Result, stage,
		cuda_Filter_rootFilterInfo->CoreLength);
	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("cudaFilter launch failed: %s\r\n", hipGetErrorString(err));
	}
	// any errors encountered during the launch.
	err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		printf("hipDeviceSynchronize returned error code %d after launching cudaFilter!\r\n", err);
	}
	
	ReleaseMutex(clsWaveFilter.hCoreMutex);

	cuda_Filter_h_Result = (FILTEDDATATYPE*)clsWaveData.FilttedBuff + clsWaveData.FilttedPos;
	err = hipMemcpy(cuda_Filter_h_Result, cuda_Filter_d_Result, cuda_Filter_result_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMemcpy cuda_Filter_h_Result failed!\r\n");
	}
	clsWaveData.FilttedPos += CUDA_FILTER_BUFF_STEP_LENGTH;
	clsWaveData.FilttedPos &= DATA_BUFFER_MASK;

	stage++;
	stage &= 0x3;
	//printf("Cuda Filtting Done.\r\n");
}